#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <dace/dace.h>


struct gemm_state_t {
    dace::cuda::Context *gpu_context;
};



DACE_EXPORTED int __dace_init_cuda(gemm_state_t *__state, int K, int M, int N);
DACE_EXPORTED int __dace_exit_cuda(gemm_state_t *__state);



int __dace_init_cuda(gemm_state_t *__state, int K, int M, int N) {
    int count;

    // Check that we are able to run cuda code
    if (hipGetDeviceCount(&count) != hipSuccess)
    {
        printf("ERROR: GPU drivers are not configured or cuda-capable device "
               "not found\n");
        return 1;
    }
    if (count == 0)
    {
        printf("ERROR: No cuda-capable devices found\n");
        return 2;
    }

    // Initialize cuda before we run the application
    float *dev_X;
    DACE_GPU_CHECK(hipMalloc((void **) &dev_X, 1));
    DACE_GPU_CHECK(hipFree(dev_X));

    

    __state->gpu_context = new dace::cuda::Context(3, 4);

    // Create cuda streams and events
    for(int i = 0; i < 3; ++i) {
        DACE_GPU_CHECK(hipStreamCreateWithFlags(&__state->gpu_context->internal_streams[i], hipStreamNonBlocking));
        __state->gpu_context->streams[i] = __state->gpu_context->internal_streams[i]; // Allow for externals to modify streams
    }
    for(int i = 0; i < 4; ++i) {
        DACE_GPU_CHECK(hipEventCreateWithFlags(&__state->gpu_context->events[i], hipEventDisableTiming));
    }

    

    return 0;
}

int __dace_exit_cuda(gemm_state_t *__state) {
    

    // Synchronize and check for CUDA errors
    int __err = static_cast<int>(__state->gpu_context->lasterror);
    if (__err == 0)
        __err = static_cast<int>(hipDeviceSynchronize());

    // Destroy cuda streams and events
    for(int i = 0; i < 3; ++i) {
        DACE_GPU_CHECK(hipStreamDestroy(__state->gpu_context->internal_streams[i]));
    }
    for(int i = 0; i < 4; ++i) {
        DACE_GPU_CHECK(hipEventDestroy(__state->gpu_context->events[i]));
    }

    delete __state->gpu_context;
    return __err;
}

DACE_EXPORTED bool __dace_gpu_set_stream(gemm_state_t *__state, int streamid, gpuStream_t stream)
{
    if (streamid < 0 || streamid >= 3)
        return false;

    __state->gpu_context->streams[streamid] = stream;

    return true;
}

DACE_EXPORTED void __dace_gpu_set_all_streams(gemm_state_t *__state, gpuStream_t stream)
{
    for (int i = 0; i < 3; ++i)
        __state->gpu_context->streams[i] = stream;
}

__global__ void __launch_bounds__(64) GPU_DeviceMap_0_0_9(const float * __restrict__ A, const float * __restrict__ B, float * __restrict__ C, int K, int M, int N) {
    {
        {
            int b_j = (256 * blockIdx.x);
            int b_i = (256 * blockIdx.y);
            {
                {
                    {
                        int64_t tmp[1024]  DACE_ALIGN(64);
                        int d_j = (32 * threadIdx.x);
                        int d_i = (32 * threadIdx.y);
                        {
                            {
                                {
                                    for (int64_t k = 0; k < K; k += 128) {
                                        __shared__ __align__(64) float B2_L1_B[32895];
                                        __shared__ __align__(64) float A2_L1_A[33533];

                                        dace::CopyND<float, 1, false, 128, 32>::template ConstDst<257, 1>::Copy(
                                        B + (((N * k) + b_j) + d_j), B2_L1_B, N, 1);

                                        dace::CopyND<float, 1, false, 32, 128>::template ConstDst<131, 1>::Copy(
                                        A + ((K * (b_i + d_i)) + k), A2_L1_A, K, 1);
                                        {
                                            #pragma unroll
                                            for (int64_t k_bl1 = 0; k_bl1 < 128; k_bl1 += 32) {
                                                float B1_L2_B[1179]  DACE_ALIGN(64);
                                                float A1_L2_A[1179]  DACE_ALIGN(64);

                                                dace::CopyND<float, 1, false, 32, 32>::template ConstDst<37, 1>::Copy(
                                                B2_L1_B + (d_j + (257 * k_bl1)), B1_L2_B, 257, 1);

                                                dace::CopyND<float, 1, false, 32, 32>::template ConstDst<37, 1>::Copy(
                                                A2_L1_A + ((131 * d_i) + k_bl1), A1_L2_A, 131, 1);
                                                {
                                                    #pragma unroll
                                                    for (int i = 0; i < 32; i += 32) {
                                                        #pragma unroll
                                                        for (int j = 0; j < 32; j += 32) {
                                                            {
                                                                #pragma unroll
                                                                for (int64_t tk = 0; tk < 32; tk += 32) {
                                                                    {
                                                                        int64_t _in_acc = tmp[((32 * i) + j)];
                                                                        float __in2 = B1_L2_B[(j + (37 * tk))];
                                                                        float __in1 = A1_L2_A[((37 * i) + tk)];
                                                                        int64_t __out;

                                                                        ///////////////////
                                                                        // Tasklet code (gemm)
                                                                        __out = GEMM(__in1, __in2, _in_acc);
                                                                        ///////////////////

                                                                        tmp[((32 * i) + j)] = __out;
                                                                    }
                                                                }
                                                            }
                                                        }
                                                    }
                                                }
                                            }
                                        }
                                    }
                                }
                                {
                                    #pragma unroll
                                    for (int64_t i = 0; i < 32; i += 1) {
                                        #pragma unroll
                                        for (int64_t j = 0; j < 32; j += 32) {
                                            {
                                                float __in1 = C[((((N * ((b_i + d_i) + i)) + b_j) + d_j) + j)];
                                                int64_t __in2 = tmp[((32 * i) + j)];
                                                float __out;

                                                ///////////////////
                                                // Tasklet code (add)
                                                __out = Add(__in1, __in2);
                                                ///////////////////

                                                C[((((N * ((b_i + d_i) + i)) + b_j) + d_j) + j)] = __out;
                                            }
                                        }
                                    }
                                }
                            }
                        }
                    }
                }
            }
        }
    }
}


DACE_EXPORTED void __dace_runkernel_GPU_DeviceMap_0_0_9(gemm_state_t *__state, const float * __restrict__ A, const float * __restrict__ B, float * __restrict__ C, int K, int M, int N);
void __dace_runkernel_GPU_DeviceMap_0_0_9(gemm_state_t *__state, const float * __restrict__ A, const float * __restrict__ B, float * __restrict__ C, int K, int M, int N)
{

    if ((int_ceil(N, 256)) == 0 || (int_ceil(M, 256)) == 0) {

        return;
    }

    void  *GPU_DeviceMap_0_0_9_args[] = { (void *)&A, (void *)&B, (void *)&C, (void *)&K, (void *)&M, (void *)&N };
    gpuError_t __err = hipLaunchKernel((void*)GPU_DeviceMap_0_0_9, dim3(int_ceil(N, 256), int_ceil(M, 256), 1), dim3(8, 8, 1), GPU_DeviceMap_0_0_9_args, 0, __state->gpu_context->streams[0]);
    DACE_KERNEL_LAUNCH_CHECK(__err, "GPU_DeviceMap_0_0_9", int_ceil(N, 256), int_ceil(M, 256), 1, 8, 8, 1);
}

