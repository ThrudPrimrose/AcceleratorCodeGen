#include <iostream>
#include <fstream>
#include <vector>
#include <hip/hip_runtime.h>
#include <mma.h>

#include <hipblas.h>

using namespace nvcuda;

// Matrix dimensions
constexpr int M = 8192;
constexpr int N = M;
constexpr int K = M;

// Paths for the binary files
std::string A_PATH = std::string("A_") + std::to_string(M) + std::string("_") + std::to_string(M) + std::string(".bin");
std::string B_PATH = std::string("B_") + std::to_string(M) + std::string("_") + std::to_string(M) + std::string(".bin");
std::string C_PATH = std::string("C_") + std::to_string(M) + std::string("_") + std::to_string(M) + std::string("_cuda_ref.bin");

#define CHECK_CUDA(call) \
do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, \
               hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

// Helper function to check cuBLAS errors
#define CHECK_CUBLAS(call) \
do { \
    hipblasStatus_t status = call; \
    if (status != HIPBLAS_STATUS_SUCCESS) { \
        printf("CUBLAS error at %s %d\n", __FILE__, __LINE__); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

void matmul_cublas_fp32(float* d_A, float* d_B, float* d_C) {
    // Create cuBLAS handle
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    // Set to row major mode
    CHECK_CUBLAS(hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH));
    CHECK_CUBLAS(hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST));

    // Prepare for row-major multiplication
    // For row-major: C = A * B becomes C' = B' * A'
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Perform matrix multiplication: C = alpha * A * B + beta * C
    // Note: cuBLAS assumes column-major order, so we transpose the operation
    // Row-major C = A * B is equivalent to column-major C' = B' * A'
    CHECK_CUBLAS(hipblasSgemm(handle,
                            HIPBLAS_OP_N,          // op(B)
                            HIPBLAS_OP_N,          // op(A)
                            N,                    // rows of op(B) and C
                            M,                    // columns of op(A) and C
                            K,                    // cols of op(B), rows of op(A)
                            &alpha,
                            d_B,                  // B matrix
                            N,                    // leading dimension of B
                            d_A,                  // A matrix
                            K,                    // leading dimension of A
                            &beta,
                            d_C,                  // C matrix
                            N));                  // leading dimension of C

}

void read_binary(const char* filename, void* data, size_t size) {
    std::ifstream file(filename, std::ios::binary);
    if (!file) {
        std::cerr << "Error opening file: " << filename << std::endl;
        exit(1);
    }
    file.read(reinterpret_cast<char*>(data), size);
    file.close();
}

// Helper function to write binary files
void write_binary(const char* filename, const void* data, size_t size) {
    std::ofstream file(filename, std::ios::binary);
    if (!file) {
        std::cerr << "Error opening file: " << filename << std::endl;
        exit(1);
    }
    file.write(reinterpret_cast<const char*>(data), size);
    file.close();
}

int main() {
    // Allocate host memory for the matrices
    std::vector<half> h_A_half(M * K);
    std::vector<half> h_B_half(K * N);
    std::vector<float> h_A(M * K);
    std::vector<float> h_B(K * N);
    std::vector<float> h_C(M * N, 0.0f);  // Initialize C to zero

    // Read the input matrices from binary files (half precision)
    read_binary(A_PATH.c_str(), h_A_half.data(), M * K * sizeof(half));
    read_binary(B_PATH.c_str(), h_B_half.data(), K * N * sizeof(half));

    // Convert half precision matrices to single precision on the host
    for (int i = 0; i < M * K; ++i) {
        h_A[i] = __half2float(h_A_half[i]);
    }
    for (int i = 0; i < K * N; ++i) {
        h_B[i] = __half2float(h_B_half[i]);
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, M * K * sizeof(float));
    hipMalloc(&d_B, K * N * sizeof(float));
    hipMalloc(&d_C, M * N * sizeof(float));

    // Copy matrices from host to device
    hipMemcpy(d_A, h_A.data(), M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), K * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_C, 0, M * N * sizeof(float));

    // Perform matrix multiplication using cuBLAS in FP32
    matmul_cublas_fp32(d_A, d_B, d_C);
    CHECK_CUDA(hipDeviceSynchronize());

    // Copy result matrix back to host
    hipMemcpy(h_C.data(), d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

    // Write the output matrix to a binary file
    write_binary(C_PATH.c_str(), h_C.data(), M * N * sizeof(float));

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    std::cout << "Matrix multiplication with cuBLAS completed and result saved to " << C_PATH.c_str() << std::endl;
    return 0;
}