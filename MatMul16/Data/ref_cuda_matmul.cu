#include <iostream>
#include <fstream>
#include <vector>
#include <hip/hip_runtime.h>
#include <mma.h>

#include <hipblas.h>

using namespace nvcuda;

// Matrix dimensions
constexpr int M_values[] = {16, 64, 4096, 8192};  // Array with different M values
constexpr int num_sizes = sizeof(M_values) / sizeof(M_values[0]);


#define CHECK_CUDA(call) \
do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, \
               hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

// Helper function to check cuBLAS errors
#define CHECK_CUBLAS(call) \
do { \
    hipblasStatus_t status = call; \
    if (status != HIPBLAS_STATUS_SUCCESS) { \
        printf("CUBLAS error at %s %d\n", __FILE__, __LINE__); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

void matmul_cublas_fp32(float* d_A, float* d_B, float* d_C, int M, int N, int K) {
    // Create cuBLAS handle
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    // Set to row major mode
    CHECK_CUBLAS(hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH));
    CHECK_CUBLAS(hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST));

    // Prepare for row-major multiplication
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Perform matrix multiplication: C = alpha * A * B + beta * C
    CHECK_CUBLAS(hipblasSgemm(handle,
                            HIPBLAS_OP_N,          // op(B)
                            HIPBLAS_OP_N,          // op(A)
                            N,                    // rows of op(B) and C
                            M,                    // columns of op(A) and C
                            K,                    // cols of op(B), rows of op(A)
                            &alpha,
                            d_B,                  // B matrix
                            N,                    // leading dimension of B
                            d_A,                  // A matrix
                            K,                    // leading dimension of A
                            &beta,
                            d_C,                  // C matrix
                            N));                  // leading dimension of C
}

void read_binary(const char* filename, void* data, size_t size) {
    std::ifstream file(filename, std::ios::binary);
    if (!file) {
        std::cerr << "Error opening file: " << filename << std::endl;
        exit(1);
    }
    file.read(reinterpret_cast<char*>(data), size);
    file.close();
}

// Helper function to write binary files
void write_binary(const char* filename, const void* data, size_t size) {
    std::ofstream file(filename, std::ios::binary);
    if (!file) {
        std::cerr << "Error opening file: " << filename << std::endl;
        exit(1);
    }
    file.write(reinterpret_cast<const char*>(data), size);
    file.close();
}

int main() {
    // Loop over different M values
    for (int i = 0; i < num_sizes; ++i) {
        int M = M_values[i];
        int N = M;  // Assuming square matrices
        int K = M;

        // Paths for the binary files
        std::string A_PATH = std::string("A") + ".bin";
        std::string B_PATH = std::string("B") + ".bin";
        std::string C_PATH = std::string("C") + "_" + std::to_string(M) + "_" + std::to_string(M) + "_cuda_ref.bin";

        // Allocate host memory for the matrices
        std::vector<half> h_A_half(M * K);
        std::vector<half> h_B_half(K * N);
        std::vector<float> h_A(M * K);
        std::vector<float> h_B(K * N);
        std::vector<float> h_C(M * N, 0.0f);  // Initialize C to zero

        // Read the input matrices from binary files (half precision)
        read_binary(A_PATH.c_str(), h_A_half.data(), M * K * sizeof(half));
        read_binary(B_PATH.c_str(), h_B_half.data(), K * N * sizeof(half));

        // Convert half precision matrices to single precision on the host
        for (int j = 0; j < M * K; ++j) {
            h_A[j] = __half2float(h_A_half[j]);
        }
        for (int j = 0; j < K * N; ++j) {
            h_B[j] = __half2float(h_B_half[j]);
        }

        // Allocate device memory
        float *d_A, *d_B, *d_C;
        hipMalloc(&d_A, M * K * sizeof(float));
        hipMalloc(&d_B, K * N * sizeof(float));
        hipMalloc(&d_C, M * N * sizeof(float));

        // Copy matrices from host to device
        hipMemcpy(d_A, h_A.data(), M * K * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B.data(), K * N * sizeof(float), hipMemcpyHostToDevice);
        hipMemset(d_C, 0, M * N * sizeof(float));

        // Perform matrix multiplication using cuBLAS in FP32
        matmul_cublas_fp32(d_A, d_B, d_C, M, N, K);
        CHECK_CUDA(hipDeviceSynchronize());

        // Copy result matrix back to host
        hipMemcpy(h_C.data(), d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

        // Write the output matrix to a binary file
        write_binary(C_PATH.c_str(), h_C.data(), M * N * sizeof(float));

        // Free device memory
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);

        std::cout << "Matrix multiplication with cuBLAS completed for M = " << M << " and result saved to " << C_PATH.c_str() << std::endl;
    }

    return 0;
}