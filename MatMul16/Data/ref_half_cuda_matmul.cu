#include <iostream>
#include <fstream>
#include <vector>
#include <hip/hip_runtime.h>
#include <mma.h>

#include <hipblas.h>

using namespace nvcuda;

// Matrix dimensions
constexpr int M = 8192;
constexpr int N = M;
constexpr int K = M;

// Paths for the binary files
std::string A_PATH = std::string("A_") + std::to_string(M) + std::string("_") + std::to_string(M) + std::string(".bin");
std::string B_PATH = std::string("B_") + std::to_string(M) + std::string("_") + std::to_string(M) + std::string(".bin");
std::string C_PATH = std::string("C_") + std::to_string(M) + std::string("_") + std::to_string(M) + std::string("_cuda_half_ref.bin");

// CUDA error check macro
#define CHECK_CUDA(call) \
do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, \
               hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

__global__ void matmul_tensor_cores(half* A, half* B, float* C, int lda, int ldb, int ldc) {
    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, 16, 16, 16, float> c_frag;

    int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
    int warpN = (blockIdx.y * blockDim.y + threadIdx.y);

    if (warpM < M / 16 && warpN < N / 16) {
        wmma::fill_fragment(c_frag, 0.0f);

        for (int i = 0; i < K; i += 16) {
            int aRow = warpM * 16;
            int aCol = i;
            int bRow = i;
            int bCol = warpN * 16;

            // Bounds checking
            if (aRow < M && aCol < K && bRow < K && bCol < N) {
                // Load the inputs
                wmma::load_matrix_sync(a_frag, A + aRow * lda + aCol, lda);
                wmma::load_matrix_sync(b_frag, B + bRow * ldb + bCol, ldb);

                // Perform the matrix multiplication
                wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
            }
        }

        wmma::store_matrix_sync(C + warpM * 16 * ldc + warpN * 16, c_frag, ldc, wmma::mem_row_major);
    }
}

// Helper functions for file I/O
void read_binary(const char* filename, void* data, size_t size) {
    std::ifstream file(filename, std::ios::binary);
    if (!file) {
        std::cerr << "Error opening file: " << filename << std::endl;
        exit(1);
    }
    file.read(reinterpret_cast<char*>(data), size);
    file.close();
}

void write_binary(const char* filename, const void* data, size_t size) {
    std::ofstream file(filename, std::ios::binary);
    if (!file) {
        std::cerr << "Error opening file: " << filename << std::endl;
        exit(1);
    }
    file.write(reinterpret_cast<const char*>(data), size);
    file.close();
}

void matmul_cublas_fp32(half* A, half* B, float* C, int m, int n, int k) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Call cublasGemmEx for tensor core-based matmul
    hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, n, m, k,
                 &alpha,
                 B, HIP_R_16F, k,
                 A, HIP_R_16F, m,
                 &beta,
                 C, HIP_R_32F, n,
                 HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);

    hipblasDestroy(handle);
}

int main() {
    // Allocate host memory for the matrices
    std::vector<half> h_A(M * K);
    std::vector<half> h_B(K * N);
    std::vector<float> h_C_tensor_cores(M * N);
    std::vector<float> h_C_cublas(M * N);

    // Read the input matrices from binary files
    read_binary(A_PATH.c_str(), h_A.data(), M * K * sizeof(half));
    read_binary(B_PATH.c_str(), h_B.data(), K * N * sizeof(half));

    // Allocate device memory
    half *d_A, *d_B;
    float *d_C;
    CHECK_CUDA(hipMalloc(&d_A, M * K * sizeof(half)));
    CHECK_CUDA(hipMalloc(&d_B, K * N * sizeof(half)));
    CHECK_CUDA(hipMalloc(&d_C, M * N * sizeof(float)));

    // Copy matrices from host to device
    CHECK_CUDA(hipMemcpy(d_A, h_A.data(), M * K * sizeof(half), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B.data(), K * N * sizeof(half), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemset(d_C, 0, M * N * sizeof(float)));

    // Launch custom kernel (Tensor Cores)
    dim3 threads(32, 16);
    dim3 blocks((N + 32 - 1) / 32, (M + 16 - 1) / 16);
    matmul_tensor_cores<<<blocks, threads>>>(d_A, d_B, d_C, K, N, N);
    CHECK_CUDA(hipDeviceSynchronize());

    // Copy result matrix back to host (Tensor Cores)
    CHECK_CUDA(hipMemcpy(h_C_tensor_cores.data(), d_C, M * N * sizeof(float), hipMemcpyDeviceToHost));

    // Run cuBLAS-based matrix multiplication
    matmul_cublas_fp32(d_A, d_B, d_C, M, N, K);

    // Copy cuBLAS result back to host
    CHECK_CUDA(hipMemcpy(h_C_cublas.data(), d_C, M * N * sizeof(float), hipMemcpyDeviceToHost));

    // Write the output matrix from tensor cores to a binary file
    write_binary(C_PATH.c_str(), h_C_cublas.data(), M * N * sizeof(float));

    // Free device memory
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));

    // Numerical verification: compare the results of the two methods
    float max_error = 0.0f;
    for (int i = 0; i < M * N; ++i) {
        float error = std::abs(h_C_tensor_cores[i] - h_C_cublas[i]);
        if (error > max_error) {
            max_error = error;
        }
    }

    std::cout << "Maximum error between Tensor Cores and cuBLAS: " << max_error << std::endl;

    return 0;
}
